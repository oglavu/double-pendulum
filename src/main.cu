#include "hip/hip_runtime.h"
#include <stdio.h>
#include <windows.h>
#include "kernel.cuh"


struct args_t {
    kernel::constants_t consts = {
        1, 1,  // l1, l2
        1, 1,  // m1, m2
        0.025, // h
        9.81,  //g
    };
    char srcFilename[100];
    char dstFilename[100];

};

void read_args(int argc, char* argv[], args_t& myArgs) {

    // read args
    if (argc < 4) {
        printf("Too few args.\n Usage: main.exe <input_file> <num_inst> <num_iter> [<options>]");
        exit(-1);
    }

    strcpy(myArgs.srcFilename, argv[1]);
    strcpy(myArgs.dstFilename, "./output.bin");

    myArgs.consts.N = atol(argv[2]);
    myArgs.consts.M = atol(argv[3]);
    
    for (int i=4; i<argc; i += 2) {
        if (argv[i][0] != '-') {
            printf("Bad cmd line args"); exit(-1);
        } 
        if (strcmp(argv[i], "-l1") == 0) {
            myArgs.consts.l1 = atof(argv[i+1]);
        } else if (strcmp(argv[i], "-l2") == 0) {
            myArgs.consts.l2 = atof(argv[i+1]);
        } else if (strcmp(argv[i], "-m1") == 0) {
            myArgs.consts.m1 = atof(argv[i+1]);
        } else if (strcmp(argv[i], "-m2") == 0) {
            myArgs.consts.m2 = atof(argv[i+1]);
        } else if (strcmp(argv[i], "-g") == 0) {
            myArgs.consts.g = atof(argv[i+1]);
        } else if (strcmp(argv[i], "-m2") == 0) {
            myArgs.consts.h = atof(argv[i+1]);
        } else if (strcmp(argv[i], "-o" ) == 0) {
            strcpy(myArgs.dstFilename, argv[i+1]);
        }

    }

}

struct mmap_t {
    HANDLE file;
    HANDLE hMap;
    size_t sz;
    void*  h_array;
};

int map_file_rd(const char* filename, struct mmap_t* map) {

    map->file = CreateFileA(
        filename, GENERIC_READ | GENERIC_WRITE, 0, 0, OPEN_EXISTING, FILE_ATTRIBUTE_NORMAL, 0
    );
    if (map->file == INVALID_HANDLE_VALUE) {
        printf("Error opening read file: %d\n", GetLastError());
        return -1;
    }

    SetFilePointer(map->file, map->sz, 0, FILE_BEGIN);
    SetEndOfFile(map->file);

    // create mapping
    map->hMap = CreateFileMapping(
        map->file, 0, PAGE_READWRITE, 0, map->sz, 0
    );
    if (!map->hMap) {
        printf("Error creating read file mapping: %d\n", GetLastError());
        CloseHandle(map->file);
        return -2;
    }

    map->h_array = MapViewOfFile(map->hMap, FILE_MAP_WRITE, 0, 0, map->sz);
    if (!map->h_array) {
        printf("Error mapping view of read file: %d\n", GetLastError());
        CloseHandle(map->hMap);
        CloseHandle(map->file);
        return -1;
    }

    return 0;
}

int map_file_wr(
    const char* filename, 
    struct mmap_t* maps, 
    const uint16_t n, 
    const size_t seg_size
) {
    HANDLE file = CreateFileA(
        filename, GENERIC_READ | GENERIC_WRITE, 0, 0, CREATE_ALWAYS, FILE_ATTRIBUTE_NORMAL, 0
    );
    if (file == INVALID_HANDLE_VALUE) {
        printf("Error opening write file: %d\n", GetLastError());
        return -1;
    }

    const uint64_t sz  = n * seg_size;
    const uint32_t szL = (uint32_t)sz;
    const uint32_t szH = (uint32_t)(sz >> 32);

    SetFilePointer(file, szL, (long*)&szH, FILE_BEGIN);
    SetEndOfFile(file);

    // create mapping
    HANDLE hMap = CreateFileMapping(
        file, 0, PAGE_READWRITE, szH, szL, 0
    );
    if (!hMap) {
        printf("Error creating write file mapping: %d\n", GetLastError());
        CloseHandle(file);
        return -2;
    }

    uint64_t offset = 0;
    for (int i=0; offset < sz; ++i) {
        maps[i].file = file;
        maps[i].hMap = hMap;
        maps[i].sz = seg_size;
        maps[i].h_array = MapViewOfFile(hMap, FILE_MAP_WRITE, (offset >> 32), (uint32_t)offset, seg_size);
        
        if (!maps[i].h_array) {
            printf("Error mapping view of write file: %d\n", GetLastError());
            for (; i>0; --i) {
                UnmapViewOfFile(maps[i].h_array); 
            }
            CloseHandle(hMap);
            CloseHandle(file);
            return -1;
        }
        offset += seg_size; 
    }

    return 0;
}

void unmap_file(struct mmap_t* maps, int n) {
    for (int i=0; i<n; i++) {
        FlushViewOfFile(maps[i].h_array, maps[i].sz);
        UnmapViewOfFile(maps[i].h_array);
    }
    CloseHandle(maps[0].hMap);
    CloseHandle(maps[0].file);
}

int main(int argc, char* argv[]) {
    // usage: main.exe <input_file> <num_instances> <num_iterations> [<options>]

    args_t myArgs;
    read_args(argc, argv, myArgs);
    kernel::set_constants(myArgs.consts);

    SYSTEM_INFO info;
    GetSystemInfo(&info);
    const uint32_t PAGE_SIZE = info.dwAllocationGranularity;
    const uint64_t SEG_SIZE  = PAGE_SIZE << 8;

    // open files
    double4 *d_initArray,
            *d_dataArray;

    size_t init_size = myArgs.consts.N * sizeof(double4);
    size_t data_size = myArgs.consts.N * myArgs.consts.M * sizeof(double4);
    uint16_t seg_count = (data_size + SEG_SIZE - 1) / SEG_SIZE;

    struct mmap_t initMap, *dataMaps = new mmap_t[seg_count];
    initMap.sz = init_size;

    if (map_file_rd(myArgs.srcFilename, &initMap) < 0) {
        return -2;
    } 
    if (map_file_wr(myArgs.dstFilename, dataMaps, seg_count, SEG_SIZE) < 0) {
        unmap_file(&initMap, 1);
        return -2;
    }

    // cuda op
    hipMalloc(&d_initArray, init_size);
    hipMalloc(&d_dataArray, seg_count * SEG_SIZE);

    hipMemcpy(d_initArray, initMap.h_array, init_size, hipMemcpyHostToDevice);

    kernel::RK4<<<1, myArgs.consts.N>>>(d_initArray, d_dataArray);

    for (int i=0; i<seg_count; ++i) {
        void* src = (void*)((uint64_t)d_dataArray + i*SEG_SIZE);
        hipMemcpy(dataMaps[i].h_array, src, SEG_SIZE, hipMemcpyDeviceToHost);
    }
    // cleanup 3 460 562 944
    hipFree(d_initArray);
    hipFree(d_dataArray);

    unmap_file(&initMap, 1);
    unmap_file(dataMaps, seg_count);

    delete[] dataMaps;

    return 0;
}