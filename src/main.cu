#include "hip/hip_runtime.h"
#include <stdio.h>
#include <iostream>
#include <windows.h>
#include "kernel.cuh"


#define gpuErrChk(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char *file, int line, bool abort=true)
{
   if (code != hipSuccess) 
   {
      fprintf(stderr,"GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
      if (abort) exit(code);
   }
}

struct args_t {
    kernel::constants_t consts = {
        1, 1,  // l1, l2
        1, 1,  // m1, m2
        0.025, // h
        9.81,  //g
    };
    char srcFilename[100];
    char dstFilename[100];

};

struct basket_t {
    struct fourlet {
        double4 data[4];
    };
    struct fourlet arr[512];
};

void read_args(int argc, char* argv[], args_t& myArgs) {

    // read args
    if (argc < 4) {
        printf("Too few args.\n Usage: main.exe <input_file> <num_inst> <num_iter> [<options>]");
        exit(-1);
    }

    strcpy(myArgs.srcFilename, argv[1]);
    strcpy(myArgs.dstFilename, "./output.bin");

    myArgs.consts.N = atol(argv[2]);
    myArgs.consts.M = atol(argv[3]);
    
    for (int i=4; i<argc; i += 2) {
        if (argv[i][0] != '-') {
            printf("Bad cmd line args"); exit(-1);
        } 
        if (strcmp(argv[i], "-l1") == 0) {
            myArgs.consts.l1 = atof(argv[i+1]);
        } else if (strcmp(argv[i], "-l2") == 0) {
            myArgs.consts.l2 = atof(argv[i+1]);
        } else if (strcmp(argv[i], "-m1") == 0) {
            myArgs.consts.m1 = atof(argv[i+1]);
        } else if (strcmp(argv[i], "-m2") == 0) {
            myArgs.consts.m2 = atof(argv[i+1]);
        } else if (strcmp(argv[i], "-g") == 0) {
            myArgs.consts.g = atof(argv[i+1]);
        } else if (strcmp(argv[i], "-m2") == 0) {
            myArgs.consts.h = atof(argv[i+1]);
        } else if (strcmp(argv[i], "-o" ) == 0) {
            strcpy(myArgs.dstFilename, argv[i+1]);
        }

    }

}

struct mmap_t {
    HANDLE file;
    HANDLE hMap;
    size_t sz;
    void*  h_array;
};

int map_file_rd(const char* filename, struct mmap_t* map) {

    map->file = CreateFileA(
        filename, GENERIC_READ | GENERIC_WRITE, 0, 0, OPEN_EXISTING, FILE_ATTRIBUTE_NORMAL, 0
    );
    if (map->file == INVALID_HANDLE_VALUE) {
        printf("Error opening read file: %d\n", GetLastError());
        return -1;
    }

    SetFilePointer(map->file, map->sz, 0, FILE_BEGIN);
    SetEndOfFile(map->file);

    // create mapping
    map->hMap = CreateFileMapping(
        map->file, 0, PAGE_READWRITE, 0, map->sz, 0
    );
    if (!map->hMap) {
        printf("Error creating read file mapping: %d\n", GetLastError());
        CloseHandle(map->file);
        return -2;
    }

    map->h_array = MapViewOfFile(map->hMap, FILE_MAP_WRITE, 0, 0, map->sz);
    if (!map->h_array) {
        printf("Error mapping view of read file: %d\n", GetLastError());
        CloseHandle(map->hMap);
        CloseHandle(map->file);
        return -1;
    }

    return 0;
}

int map_file_wr(
    const char* filename, 
    struct mmap_t* maps, 
    const uint16_t n, 
    const size_t seg_size
) {
    HANDLE file = CreateFileA(
        filename, GENERIC_READ | GENERIC_WRITE, 0, 0, CREATE_ALWAYS, FILE_ATTRIBUTE_NORMAL, 0
    );
    if (file == INVALID_HANDLE_VALUE) {
        printf("Error opening write file: %d\n", GetLastError());
        return -1;
    }

    const uint64_t sz  = n * seg_size;
    const uint32_t szL = (uint32_t)sz;
    const uint32_t szH = (uint32_t)(sz >> 32);

    SetFilePointer(file, szL, (long*)&szH, FILE_BEGIN);
    SetEndOfFile(file);

    // create mapping
    HANDLE hMap = CreateFileMapping(
        file, 0, PAGE_READWRITE, szH, szL, 0
    );
    if (!hMap) {
        printf("Error creating write file mapping: %d\n", GetLastError());
        CloseHandle(file);
        return -2;
    }

    uint64_t offset = 0;
    for (int i=0; offset < sz; ++i) {
        maps[i].file = file;
        maps[i].hMap = hMap;
        maps[i].sz = seg_size;
        maps[i].h_array = MapViewOfFile(hMap, FILE_MAP_WRITE, (offset >> 32), (uint32_t)offset, seg_size);
        
        if (!maps[i].h_array) {
            printf("Error mapping view of write file: %d\n", GetLastError());
            for (; i>0; --i) {
                UnmapViewOfFile(maps[i].h_array); 
            }
            CloseHandle(hMap);
            CloseHandle(file);
            return -1;
        }
        offset += seg_size; 
    }

    return 0;
}

void unmap_file(struct mmap_t* maps, int n) {
    for (int i=0; i<n; i++) {
        FlushViewOfFile(maps[i].h_array, maps[i].sz);
        UnmapViewOfFile(maps[i].h_array);
    }
    CloseHandle(maps[0].hMap);
    CloseHandle(maps[0].file);
}

int main(int argc, char* argv[]) {
    // usage: main.exe <input_file> <num_instances> <num_iterations> [<options>]

    args_t myArgs;
    read_args(argc, argv, myArgs);

    SYSTEM_INFO info;
    GetSystemInfo(&info);
    const uint32_t PAGE_SIZE = info.dwAllocationGranularity;
    const uint64_t SEG_SIZE  = PAGE_SIZE << 8;

    size_t free_size, _;
    gpuErrChk( hipMemGetInfo(&free_size, &_) );

    // open files
    double4 *d_initArray,
            *d_dataArray;

    size_t init_size = myArgs.consts.N * sizeof(double4);
    size_t data_size = myArgs.consts.N * myArgs.consts.M * sizeof(double4);
    uint16_t seg_count = (data_size + SEG_SIZE - 1) / SEG_SIZE;

    struct mmap_t initMap, *dataMaps = new mmap_t[seg_count];
    initMap.sz = init_size;

    if (map_file_rd(myArgs.srcFilename, &initMap) < 0) {
        return -2;
    } 
    if (map_file_wr(myArgs.dstFilename, dataMaps, seg_count, SEG_SIZE) < 0) {
        unmap_file(&initMap, 1);
        return -2;
    }

    // cuda op

    gpuErrChk( hipMalloc(&d_initArray, init_size) );

    size_t turn_size = 1 << 30; // free_size & (-SEG_SIZE);
    // while( hipMalloc(&d_dataArray, turn_size) != hipSuccess ) {
    //     // allocate whole number of SEGs
    //     turn_size = (turn_size >> 1) & (-SEG_SIZE);
    //     if (turn_size < SEG_SIZE) {
    //         printf("Not possible to allocate even one SEG\n");
    //     }
    // }
    gpuErrChk( hipMalloc(&d_dataArray, turn_size) );
    const uint32_t sg_per_turn = turn_size / SEG_SIZE;
    const uint32_t bs_per_sg = SEG_SIZE / sizeof(basket_t);

    myArgs.consts.M = sg_per_turn * bs_per_sg;
    
    //printf("sg_per_turn: %lu\n bs_per_sg: %lu\n M: %lu\n SEG_SIZE: 0x%llx\n", sg_per_turn, bs_per_sg, myArgs.consts.M, SEG_SIZE);
    kernel::set_constants(myArgs.consts);
    for (int j=0, i=0; i<seg_count; ++i) {
        
        if (i % sg_per_turn == 0) {
            double4* tail_basket = (double4*)& ((basket_t*)dataMaps[i-1].h_array)[bs_per_sg-2];
            double4* init_basket = (j == 0 ? (double4*)initMap.h_array : tail_basket);

            gpuErrChk( hipMemcpy(d_initArray, init_basket, init_size, hipMemcpyHostToDevice) );

            kernel::RK4<<<1, myArgs.consts.N>>>(d_initArray, d_dataArray);

            hipError_t err = hipGetLastError();
            if (err != hipSuccess) {
                std::cout << "Kernel launch failed: " << hipGetErrorString(err) << std::endl;
            }
            hipDeviceSynchronize();
            ++j;
        }

        // void* src = d_dataArray;
        void* src = (void*)((uint64_t)d_dataArray + (i%sg_per_turn)*SEG_SIZE);
        void* dst = dataMaps[i].h_array;
        
        //printf("i: %d, src: 0x%llx, dst: 0x%llx\n", i, (uint64_t)src, (uint64_t)dst);
        //std::memset(dst, 1, SEG_SIZE);
        //gpuErrChk( hipMemset(src, 1, SEG_SIZE) );
        gpuErrChk( hipMemcpy(dst, src, (size_t)SEG_SIZE, hipMemcpyDeviceToHost));
    }

    // cleanup
    hipFree(d_initArray);
    hipFree(d_dataArray);

    unmap_file(&initMap, 1);
    unmap_file(dataMaps, seg_count);

    delete[] dataMaps;

    return 0;
}